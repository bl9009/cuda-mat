#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "..\include\matrix.cuh"

typedef matrix_t h_matrix_t; // host matrix type
typedef matrix_t d_matrix_t; // device matrix type
typedef vector_t h_vector_t;
typedef vector_t d_vector_t;

#ifdef __cplusplus
extern "C" {
#endif

// private interface
d_matrix_t load_device(matrix_t A);
void unload_device(d_matrix_t d_mat);
void fetch_unload_device(d_matrix_t d_mat, h_matrix_t* h_mat);

#ifdef __cplusplus
}
#endif

// public interface
static matrix_t construct(size_t rows, size_t cols)
{
    matrix_t mat = { 0, 0, rows, cols };

    mat.pitch = cols * sizeof(double);
    mat.data = (double*)malloc(mat.pitch * rows);

    return mat;
}

matrix_t zeros(size_t rows, size_t cols)
{
    matrix_t A = construct(rows, cols);

    for (int i = 0; i < rows * cols; ++i) {
        A.data[i] = 0.0;
    }

    return A;
}

__device__ double* cell_device(d_matrix_t A, size_t row, size_t col)
{
    double* r = (double*)((char*)A.data + row * A.pitch);
    return &r[col];
}

error_t cell(matrix_t A, size_t row, size_t col, double** value)
{
    if (row >= A.rows || col >= A.cols) {
        return OUT_OF_BOUNDS;
    }

    double* r = (double*)((char*) A.data + row * A.pitch);
    *value = &r[col];

    return OK;
}

error_t from_seq(double* seq, matrix_t* A)
{
    for (int i = 0; i < A->rows; ++i) {
        for (int j = 0; j < A->cols; ++j) {
            double* tmp = NULL;

            if (cell(*A, i, j, &tmp) == OUT_OF_BOUNDS) {
                return OUT_OF_BOUNDS;
            }

            *tmp = seq[i * A->cols + j];
            //*cell(*A, i, j) = seq[i * A->cols + j];
        }
    }

    return OK;
}

shape_t mult_result_shape(matrix_t A, matrix_t B)
{
    shape_t shape = { A.rows, B.cols };

    return shape;
}

__global__ void mult_kernel(d_matrix_t d_A, d_matrix_t d_B, d_matrix_t d_C)
{
    size_t row = threadIdx.x;
    size_t col = threadIdx.y;

    double* c = cell_device(d_C, row, col);

    double sum = 0.0;

    for (int k = 0; k < d_A.cols; ++k) {
        sum += *cell_device(d_A, row, k) * *cell_device(d_B, k, col);
    }

    *c = sum;

}

error_t mult(matrix_t A, matrix_t B, matrix_t* C)
{
    h_matrix_t h_C = *C;

    size_t res_rows = A.rows;
    size_t res_cols = B.cols;

    if (res_rows != res_cols ||
        res_rows != h_C.rows ||
        res_cols != h_C.cols) {
        return SHAPE_MISMATCH;
        }

    d_matrix_t d_A = load_device(A);
    d_matrix_t d_B = load_device(B);
    d_matrix_t d_C = load_device(h_C);

    mult_kernel<<<dim3(1,1,1), dim3((unsigned)d_C.rows, (unsigned)d_C.cols,1)>>>(d_A, d_B, d_C);

    hipDeviceSynchronize();

    unload_device(d_A);
    unload_device(d_B);

	fetch_unload_device(d_C, &h_C);

    return OK;
}

__global__ void transpose_kernel(d_matrix_t d_A, d_matrix_t d_A_t)
{
    size_t row = threadIdx.x;
    size_t col = threadIdx.y;

    *cell_device(d_A_t, row, col) = *cell_device(d_A, col, row);
}

shape_t transpose_result_shape(matrix_t A)
{
    shape_t shape = { A.cols, A.rows };

    return shape;
}

error_t transpose(matrix_t A, matrix_t* A_t)
{
    h_matrix_t h_A = A;
    h_matrix_t h_A_t = *A_t;

    d_matrix_t d_A = load_device(h_A);
    d_matrix_t d_A_t = load_device(h_A_t);

    transpose_kernel<<<dim3(1,1,1),dim3((unsigned)d_A_t.rows, (unsigned)d_A_t.cols, 1)>>>(d_A, d_A_t);

    unload_device(d_A);
	fetch_unload_device(d_A_t, &h_A_t);

    return OK;
}

void destruct(matrix_t A)
{
	free(A.data);
}

void print(matrix_t mat)
{
    for (int i = 0; i < mat.rows; ++i) {
        for (int j = 0; j < mat.cols; ++j) {
            double* tmp = NULL;

            if (cell(mat, i, j, &tmp) == OUT_OF_BOUNDS) {
                printf("index out of bounds!");

                return;
            }

            printf("[%f] ", *tmp);
        }

        printf("\n");
    }

    printf("\n");
}

d_matrix_t load_device(h_matrix_t h_mat)
{
    d_matrix_t d_mat = { 0, 0, h_mat.rows, h_mat.cols };

    hipMallocPitch(&d_mat.data,
        &d_mat.pitch,
        sizeof(double) * h_mat.cols,
        h_mat.rows);

    hipMemcpy2D(d_mat.data,
        d_mat.pitch,
        h_mat.data,
        h_mat.pitch,
        sizeof(double) * h_mat.cols,
        h_mat.rows,
        hipMemcpyHostToDevice);

    return d_mat;
}

void unload_device(d_matrix_t d_mat)
{
    hipFree(d_mat.data);
}

void fetch_unload_device(d_matrix_t d_mat, h_matrix_t* h_mat)
{
    hipMemcpy2D((*h_mat).data,
        (*h_mat).pitch,
        d_mat.data,
        d_mat.pitch,
        sizeof(double) * d_mat.cols,
        d_mat.rows,
        hipMemcpyDeviceToHost);

    hipFree(d_mat.data);
}

#define construct NOT_ALLOWED
#define load_device NOT_ALLOWED
#define unload_device NOT_ALLOWED
#define mult_kernel NOT_ALLOWED
#define transpose_kernel NOT_ALLOWED
