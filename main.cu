#include <stdio.h>
#include <stdlib.h>

#include "src/matrix.cuh"

int main()
{
    double seqA[6] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0 };
    unsigned m = 2;
    unsigned n = 3;

    double seqB[6] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0 };
    unsigned p = 3;
    unsigned q = 2;

    matrix_t A = from_seq(seqA, m, n);
    matrix_t B = from_seq(seqB, p, q);

    matrix_t C = mult(A, B);

    print(C);

#ifdef __HIPCC__
    matrix_t D = mult2(A, B);
    print(D);
#endif

    clear(A);
    clear(B);
    clear(C);
    printf("here12");
#ifdef __HIPCC__
    clear(D);
    printf("here13");
#endif

    return 0;
}


/*#include <iostream>

#include "src/matrix.cuh"

int main()
{
    std::cout << "my mat lib" << std::endl;

    cudamat::MatrixData dataA{ { 1, 2, 3 }, { 4, 5, 6 }, { 7, 8, 9 }, { 10, 11, 12 }, { 13, 14, 15 } };
    cudamat::MatrixData dataB{ { 1, 2, 7 }, { 3, 4, 8 }, { 5, 6, 9 } };

    cudamat::Matrix A = cudamat::Matrix::create(dataA);
    cudamat::Matrix B = cudamat::Matrix::create(dataB);

    std::cout << "A shape - rows: " << A.shape().rows << ", cols: " << A.shape().cols << std::endl;
    std::cout << "B shape - rows: " << B.shape().rows << ", cols: " << B.shape().cols << std::endl;

    cudamat::Matrix C = A.mult(B);

    std::cout << "C shape - rows: " << C.shape().rows << ", cols: " << C.shape().cols << std::endl;

    cudamat::Matrix::print(C);

    return 0;
}*/
